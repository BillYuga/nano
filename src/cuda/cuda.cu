#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>

static void HandleError(hipError_t err, const char*file, int line)
{
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
                exit(EXIT_FAILURE);
        }
}

#define CUDA_HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

namespace nano
{
        const cuda::manager_t& cuda::manager_t::instance()
        {
                static const cuda::manager_t the_instance;
                return the_instance;
        }

        cuda::manager_t::manager_t() :
                m_devices(0)
        {
                CUDA_HANDLE_ERROR(hipGetDeviceCount(&m_devices));

                for (int device = 0; device < m_devices; device ++)
                {
                        CUDA_HANDLE_ERROR(hipGetDeviceProperties(&m_properties[device], device));
                }
        }

        bool cuda::manager_t::print_info() const
        {
                const int count = count_devices();
                for (int i = 0; i < count; i ++)
                {
                        const hipDeviceProp_t prop = get_device_properties(i);

                        printf("CUDA device [%d/%d]: name = %s\n", i + 1, count, prop.name);
                        printf("CUDA device [%d/%d]: compute capability = %d.%d\n", i + 1, count, prop.major, prop.minor);
                        printf("CUDA device [%d/%d]: clock rate = %d\n", i + 1, count, prop.clockRate);
                        printf("CUDA device [%d/%d]: global mem = %ld\n", i + 1, count, prop.totalGlobalMem);
                        printf("CUDA device [%d/%d]: constant mem = %ld\n", i + 1, count, prop.totalConstMem);
                        printf("CUDA device [%d/%d]: mem pitch = %ld\n", i + 1, count, prop.memPitch);
                        printf("CUDA device [%d/%d]: texture alignment = %ld\n", i + 1, count, prop.textureAlignment);
                        printf("CUDA device [%d/%d]: multiprocessor count = %d\n", i + 1, count, prop.multiProcessorCount);
                        printf("CUDA device [%d/%d]: shared mem per mp = %ld\n", i + 1, count, prop.sharedMemPerBlock);
                        printf("CUDA device [%d/%d]: registers per mp = %d\n", i + 1, count, prop.regsPerBlock);
                        printf("CUDA device [%d/%d]: threads in warp = %d\n", i + 1, count, prop.warpSize);
                        printf("CUDA device [%d/%d]: max threads per block = %d\n", i + 1, count, prop.maxThreadsPerBlock);
                        printf("CUDA device [%d/%d]: max thread dimensions = (%d, %d, %d)\n", i + 1, count,
                               prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
                        printf("CUDA device [%d/%d]: max grid dimensions = (%d, %d, %d)\n", i + 1, count,
                               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
                        printf("CUDA device [%d/%d]: asynchronous engines = %d\n", i + 1, count, prop.asyncEngineCount);
                        printf("\n");
                }

                return true;
        }

        int cuda::count_devices()
        {
                return manager_t::instance().count_devices();
        }

        hipDeviceProp_t cuda::get_device_properties(int device)
        {
                return manager_t::instance().get_device_properties(device);
        }

        bool cuda::print_info()
        {
                return manager_t::instance().print_info();
        }

        static int round_divide(int size, int divisor)
        {
                return (size + divisor - 1) / divisor;
        }

        dim3 cuda::make_blocks1d(int size, int device)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                const int threads = prop.maxThreadsPerBlock;

                return dim3(round_divide(size, threads));
        }

        dim3 cuda::make_threads1d(int, int device)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                const int threads = prop.maxThreadsPerBlock;

                return dim3(threads);
        }

        dim3 cuda::make_blocks2d(int rows, int cols, int device)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                const int threads = int(sqrt(prop.maxThreadsPerBlock));

                return dim3(round_divide(cols, threads),
                            round_divide(rows, threads));
        }

        dim3 cuda::make_threads2d(int, int, int device)
        {
                const hipDeviceProp_t prop = cuda::get_device_properties(device);
                const int threads = int(sqrt(prop.maxThreadsPerBlock));

                return dim3(threads,
                            threads);
        }
}

#include "hip/hip_runtime.h"
#include "conv2d.h"
#include "hip/hip_runtime.h"

template
<
        typename tscalar
>
__global__ void kernel_conv2d(
        const tscalar* idata,
        const tscalar* kdata, int krows, int kcols,
        tscalar* odata, int orows, int ocols)
{
        const int c = threadIdx.x + blockIdx.x * blockDim.x;
        const int r = threadIdx.y + blockIdx.y * blockDim.y;

        if (r < orows && c < ocols)
        {
                const int icols = ocols + kcols - 1;

                tscalar sum = 0;
                for (int kr = 0; kr < krows; kr ++)
                {
                        for (int kc = 0; kc < kcols; kc ++)
                        {
                                sum += idata[(r + kr) * icols + (c + kc)] * kdata[kr * kcols + kc];
                        }
                }

                odata[r * ocols + c] = sum;
        }
}

template
<
        typename tscalar
>
__global__ void kernel_iconv2d(
        const tscalar* odata,
        const tscalar* kdata, int krows, int kcols,
        tscalar* idata, int irows, int icols)
{
        const int c = threadIdx.x + blockIdx.x * blockDim.x;
        const int r = threadIdx.y + blockIdx.y * blockDim.y;

        if (r < irows && c < icols)
        {
                const int orows = irows - krows + 1;
                const int ocols = icols - kcols + 1;

                const int krmin = max(0,     r - orows + 1);
                const int krmax = min(krows, r + 1);

                const int kcmin = max(0,     c - ocols + 1);
                const int kcmax = min(kcols, c + 1);

                tscalar sum = 0;
                for (int kr = krmin; kr < krmax; kr ++)
                {
                        for (int kc = kcmin; kc < kcmax; kc ++)
                        {
                                sum += odata[(r - kr) * ocols + (c - kc)] * kdata[kr * kcols + kc];
                        }
                }

                idata[r * icols + c] = sum;
        }
}

namespace nano
{
        template
        <
                typename tscalar
        >
        static bool cuda_conv2d(
                const cuda::matrix_t<tscalar>& idata,
                const cuda::matrix_t<tscalar>& kdata,
                cuda::matrix_t<tscalar>& odata,
                int device)
        {
                if (    odata.rows() + kdata.rows() != idata.rows() + 1 ||
                        odata.cols() + kdata.cols() != idata.cols() + 1)
                {
                        return false;
                }

                else
                {
                        const dim3 bsize = cuda::make_blocks2d(odata.rows(), odata.cols(), device);
                        const dim3 tsize = cuda::make_threads2d(odata.rows(), odata.cols(), device);

                        kernel_conv2d<<<bsize, tsize>>>(
                                idata.data(),
                                kdata.data(), kdata.rows(), kdata.cols(),
                                odata.data(), odata.rows(), odata.cols());

                        return hipGetLastError() == hipSuccess;
                }
        }

        bool cuda::conv2d(const imatrix_t& idata, const imatrix_t& kdata, imatrix_t& odata, int device)
        {
                return cuda_conv2d(idata, kdata, odata, device);
        }

        bool cuda::conv2d(const fmatrix_t& idata, const fmatrix_t& kdata, fmatrix_t& odata, int device)
        {
                return cuda_conv2d(idata, kdata, odata, device);
        }

        bool cuda::conv2d(const dmatrix_t& idata, const dmatrix_t& kdata, dmatrix_t& odata, int device)
        {
                return cuda_conv2d(idata, kdata, odata, device);
        }

        template
        <
                typename tscalar
        >
        static bool cuda_iconv2d(
                const cuda::matrix_t<tscalar>& odata,
                const cuda::matrix_t<tscalar>& kdata,
                cuda::matrix_t<tscalar>& idata,
                int device)
        {
                if (    odata.rows() + kdata.rows() != idata.rows() + 1 ||
                        odata.cols() + kdata.cols() != idata.cols() + 1)
                {
                        return false;
                }

                else
                {
                        const dim3 bsize = cuda::make_blocks2d(idata.rows(), idata.cols(), device);
                        const dim3 tsize = cuda::make_threads2d(idata.rows(), idata.cols(), device);

                        kernel_iconv2d<<<bsize, tsize>>>(
                                odata.data(),
                                kdata.data(), kdata.rows(), kdata.cols(),
                                idata.data(), idata.rows(), idata.cols());

                        return hipGetLastError() == hipSuccess;
                }
        }

        bool cuda::iconv2d(const imatrix_t& odata, const imatrix_t& kdata, imatrix_t& idata, int device)
        {
                return cuda_iconv2d(odata, kdata, idata, device);
        }

        bool cuda::iconv2d(const fmatrix_t& odata, const fmatrix_t& kdata, fmatrix_t& idata, int device)
        {
                return cuda_iconv2d(odata, kdata, idata, device);
        }

        bool cuda::iconv2d(const dmatrix_t& odata, const dmatrix_t& kdata, dmatrix_t& idata, int device)
        {
                return cuda_iconv2d(odata, kdata, idata, device);
        }
}
